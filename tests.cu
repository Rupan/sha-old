#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "sha1.h"
#include "sha2.h"

#define N 37426 /* the (arbitrary) number of hashes we want to calculate */
#define THREAD_COUNT 128

__device__ const unsigned char *m = "Goodbye, cruel world!";

__global__ void kernel_sha1(unsigned char *hval) {
  sha1_ctx ctx[1];
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while(tid < N) {
    sha1_begin(ctx);
    sha1_hash(m, 21UL, ctx);
    sha1_end(hval+tid*SHA1_DIGEST_SIZE, ctx);
    tid += blockDim.x * gridDim.x;
  }
}

__global__ void kernel_sha2(unsigned char *hval) {
  sha256_ctx ctx[1];
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while(tid < N) {
    sha256_begin(ctx);
    sha256_hash(m, 21UL, ctx);
    sha256_end(hval+tid*SHA256_DIGEST_SIZE, ctx);
    tid += blockDim.x * gridDim.x;
  }
}

int main( void ) {
  int device_count;
  unsigned int i, j, blocks, offset;
  float sha1_elapsed, sha256_elapsed;
  hipError_t err;
  hipEvent_t start, stop;
  hipDeviceProp_t prop;
  unsigned char *host_hval;
  unsigned char *device_hval;

  err = hipEventCreate(&start);
  if(err != hipSuccess) { }
  err = hipEventCreate(&stop);
  if(err != hipSuccess) { }

  err = hipGetDeviceCount(&device_count);
  if(err != hipSuccess) { }
  if(device_count == 0) {
    printf("Device not found.\n");
    return 1;
  }
  // hard-code device 0 for now
  err = hipGetDeviceProperties(&prop, 0);
  if(err != hipSuccess) {
    printf("Failed to get device properties.\n");
    return 1;
  }
  blocks = prop.multiProcessorCount * 6;

  /* allocate enough memory for the largest output required */
  host_hval = (unsigned char *)malloc(N*SHA256_DIGEST_SIZE);
  if(host_hval == NULL) {
    printf("Failed to allocate host memory.\n");
    return 1;
  }
  err = hipMalloc((void**)&device_hval, N*SHA256_DIGEST_SIZE);
  if(err != hipSuccess) {
    printf("Failed to allocate device memory.\n");
    err = hipFree(device_hval);
    if(err != hipSuccess) {
      printf("Failed to free device memory.\n");
    }
    free(host_hval);
    return 1;
  }

  /* test SHA1 */
  err = hipMemset(device_hval, 0, N*SHA256_DIGEST_SIZE);
  if(err != hipSuccess) {
    printf("SHA1: Failed to initialize device memory.\n");
    err = hipFree(device_hval);
    if(err != hipSuccess) { 
      printf("SHA1: Failed to free device memory.\n");
    }
    free(host_hval);
    return 1;
  }
  err = hipEventRecord(start, 0);
  if(err != hipSuccess) { }
  kernel_sha1<<<blocks,THREAD_COUNT>>>(device_hval);
  err = hipEventRecord(stop, 0);
  if(err != hipSuccess) { }
  err = hipEventSynchronize(stop);
  if(err != hipSuccess) { }
  err = hipEventElapsedTime(&sha1_elapsed, start, stop);
  if(err != hipSuccess) { }
  err = hipMemcpy(host_hval, device_hval, N*SHA1_DIGEST_SIZE, hipMemcpyDeviceToHost);
  if(err != hipSuccess) {
    printf("SHA1: hipMemcpy failed.\n");
    err = hipFree(device_hval);
    if(err != hipSuccess) {
      printf("SHA1: Failed to free device memory.\n");
    }
    free(host_hval);
    return 1;
  }
  /* should print: f99ec870820d0832c9e04e9e4a0e40fb5038636c */
  for(i=0; i<N; i++) {
    offset = i * SHA1_DIGEST_SIZE;
    for(j=0; j<SHA1_DIGEST_SIZE; j++)
      printf("%02X", host_hval[offset+j]);
    printf("\n");
  }

  /* test SHA2 */
  err = hipMemset(device_hval, 0, N*SHA256_DIGEST_SIZE);
  if(err != hipSuccess) {
    printf("SHA256: Failed to initialize device memory.\n");
    err = hipFree(device_hval);
    if(err != hipSuccess) {
      printf("SHA256: Failed to free device memory.\n");
    }
    free(host_hval);
    return 1;
  }
  err = hipEventRecord(start, 0);
  if(err != hipSuccess) { }
  kernel_sha2<<<blocks,THREAD_COUNT>>>(device_hval);
  hipEventRecord(stop, 0);
  if(err != hipSuccess) { }
  err = hipEventSynchronize(stop);
  if(err != hipSuccess) { }
  err = hipEventElapsedTime(&sha256_elapsed, start, stop);
  if(err != hipSuccess) { }
  err = hipMemcpy(host_hval, device_hval, N*SHA256_DIGEST_SIZE, hipMemcpyDeviceToHost);
  if(err != hipSuccess) {
    printf("SHA256: hipMemcpy failed.\n");
    err = hipFree(device_hval);
    if(err != hipSuccess) {
      printf("SHA256: Failed to free device memory.\n");
    }
    free(host_hval);
    return 1;
  }
  /* should print: 469c24f94970733aa9d3c18ba88b816a5572cdc86286c30107e3ffcb9ef88e05 */
  for(i=0; i<N; i++) {
    offset = i * SHA256_DIGEST_SIZE;
    for(j=0; j<SHA256_DIGEST_SIZE; j++)
      printf("%02X", host_hval[offset+j]);
    printf("\n");
  }

  printf("Timers: SHA1 = %5.3f ms; SHA256 = %5.3f ms\n", sha1_elapsed, sha256_elapsed);

  /* clean up */
  err = hipFree(device_hval);
  if(err != hipSuccess) {
    printf("FINAL: Failed to free device memory.\n");
  }
  err = hipEventDestroy(start);
  if(err != hipSuccess)
    printf("FINAL: Unable to destroy start event");
  err = hipEventDestroy(stop);
  if(err != hipSuccess)
    printf("FINAL: Unable to destroy stop event");
  device_hval = NULL;
  free(host_hval);
  host_hval = NULL;

  return 0;
}
